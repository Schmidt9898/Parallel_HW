#include "hip/hip_runtime.h"
/* Copyright (c) 2012, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <math.h>
#include <string.h>
#include <stdlib.h>
#include <omp.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <thrust/reduce.h>
#include <thrust/device_ptr.h>
#include <chrono>


class Timer {
public:
	std::chrono::time_point<std::chrono::high_resolution_clock> start, end;
	std::chrono::duration<float> duration;
	std::string name;

	Timer(std::string name_ = "Timer") : name(name_){
        start = std::chrono::high_resolution_clock::now();
    }
	~Timer() {
		end = std::chrono::high_resolution_clock::now();
		duration = end - start;
		std::cout << name << " took: " << duration.count() << "\n";
	}
	

};




//This is a little wrapper that checks for error codes returned by CUDA API calls
#define cudaCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void my_stencil_kernel(double *A, double *Anew, int imax, int jmax, double *d_x) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if (i >= 1 && i < imax+1 && j >= 1 && j < jmax+1) {
        Anew[(j)*(imax+2)+i] = 0.25f * ( A[(j)*(imax+2)+i+1] + A[(j)*(imax+2)+i-1]
                            + A[(j-1)*(imax+2)+i] + A[(j+1)*(imax+2)+i]);
    
        d_x[(j)*(imax+2)+i] = fabs(Anew[(j)*(imax+2)+i]-A[(j)*(imax+2)+i]);
    }
}

__global__ void my_copy_kernel(double *A, double *Anew, int imax, int jmax) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if (i >= 1 && i < imax+1 && j >= 1 && j < jmax+1)
        A[(j)*(imax+2)+i] = Anew[(j)*(imax+2)+i];    
}



__global__ void error_reduce_kernel(double *d_x, int xmax, int ymax,double * error)
{
    // dynamically allocated shared memory
    extern  __shared__  double temp[];
    
    int tid = threadIdx.x;
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    // first, each thread loads data into shared memory
    if (y<ymax && x<xmax)
    {
        temp[tid]=d_x[(y)*(xmax+2)+x];
        //printf("%f ",d_x[(y)*(xmax+2)+x]);
    }
    else
    {
        temp[tid]=0;
    }
    // next, we perform binary tree reduction

    for (int d = blockDim.x>>1; d > 0; d >>= 1) {
      __syncthreads();  // ensure previous step completed 
      if (tid<d)  temp[tid] = fmax(temp[tid],temp[tid+d]);
    }

    // finally, first thread puts result into global memory

    if (tid==0) *error = temp[0];
}



int main(int argc, char** argv)
{
    //Size along y
    int jmax = 4094;
    //Size along x
    int imax = 4094;
    int iter_max = 1000;
    
    const double pi  = 2.0 * asin(1.0);
    const double tol = 1.0e-5;
    double error     = 1.0;

    double *A;
    double *Anew;
    double *y0;

    A    = (double *)malloc((imax+2) * (jmax+2) * sizeof(double));
    Anew = (double *)malloc((imax+2) * (jmax+2) * sizeof(double));
    y0   = (double *)malloc((imax+2) * sizeof(double));

    memset(A, 0, (imax+2) * (jmax+2) * sizeof(double));
    
    // set boundary conditions
    for (int i = 0; i < imax+2; i++)
      A[(0)*(imax+2)+i]   = 0.0;

    for (int i = 0; i < imax+2; i++)
      A[(jmax+1)*(imax+2)+i] = 0.0;
    
    for (int j = 0; j < jmax+2; j++)
    {
        y0[j] = sin(pi * j / (jmax+1));
        A[(j)*(imax+2)+0] = y0[j];
    }

    for (int j = 0; j < imax+2; j++)
    {
        y0[j] = sin(pi * j/ (jmax+1));
        A[(j)*(imax+2)+imax+1] = y0[j]*exp(-pi);
    }
    
    printf("Jacobi relaxation Calculation: %d x %d mesh\n", imax+2, jmax+2);
    {

        Timer("GPU");
        int iter = 0;
    
    for (int i = 1; i < imax+2; i++)
       Anew[(0)*(imax+2)+i]   = 0.0;

    for (int i = 1; i < imax+2; i++)
       Anew[(jmax+1)*(imax+2)+i] = 0.0;

    for (int j = 1; j < jmax+2; j++)
        Anew[(j)*(imax+2)+0]   = y0[j];

    for (int j = 1; j < jmax+2; j++)
        Anew[(j)*(imax+2)+jmax+1] = y0[j]*expf(-pi);
    
    double *d_A;
    double *d_Anew;
    double *d_x;
    double *d_error;
    cudaCheck(hipMalloc(&d_error,sizeof(double)));
    cudaCheck(hipMalloc(&d_A, (imax+2) * (jmax+2) * sizeof(double)));
    cudaCheck(hipMemcpy( d_A, A, (imax+2) * (jmax+2) * sizeof(double), hipMemcpyHostToDevice ));
    cudaCheck(hipMalloc(&d_Anew, (imax+2) * (jmax+2) * sizeof(double)));
    cudaCheck(hipMemcpy( d_Anew, Anew, (imax+2) * (jmax+2) * sizeof(double), hipMemcpyHostToDevice ));
    cudaCheck(hipMalloc(&d_x, (imax+2) * (jmax+2) * sizeof(double)));
    cudaCheck(hipMemset(d_x, 0, (imax+2) * (jmax+2) * sizeof(double)));

    while ( error > tol && iter < iter_max )
    {
        error =0.0;

        dim3 block(16,4);
        dim3 grid((imax+2-1)/16+1, (jmax+2-1)/4+1);
        my_stencil_kernel<<<grid,block>>>(d_A, d_Anew, imax, jmax, d_x);
        hipDeviceSynchronize();
        error_reduce_kernel<<<grid,block,sizeof(double)*1024>>>(d_x,imax, jmax, d_error);

        cudaCheck(hipMemcpy( &error, d_error, sizeof(double), hipMemcpyDeviceToHost ));
        
        //error = thrust::reduce(
            //        thrust::device_ptr<double>(d_x),
            //        thrust::device_ptr<double>(d_x+(imax+2) * (jmax+2)),
            //        0.0,thrust::maximum<double>());
            
        //No stencil accesses to Anew, no halo exchange necessary
        my_copy_kernel<<<grid,block>>>(d_A, d_Anew, imax, jmax);
        hipDeviceSynchronize();

        if(iter % 100 == 0) printf("%5d, %0.6f\n", iter, error);
        
        iter++;
    }
    
    //double runtime = omp_get_wtime()-t1;
    }
    
    //printf(" total: %f s\n", runtime);
}

